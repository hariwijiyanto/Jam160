#include <hip/hip_runtime.h>
#include <stdint.h>
#include <string.h>

// --- Kode dari secp256k1.cuh ---
#define BIGINT_WORDS 8

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error in %s at line %d: %s\\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

struct BigInt {
    uint32_t data[BIGINT_WORDS];
};

struct ECPoint {
    BigInt x, y;
    bool infinity;
};

struct ECPointJac {
    BigInt X, Y, Z;
    bool infinity;
};

__constant__ BigInt const_p;
__constant__ ECPointJac const_G_jacobian;
__constant__ BigInt const_n;

// Precomputation table untuk 2^i * G (for i=0 to 255)
__device__ ECPointJac const_G_table[256];

__host__ __device__ __forceinline__ void init_bigint(BigInt *x, uint32_t val) {
    x->data[0] = val;
    for (int i = 1; i < BIGINT_WORDS; i++) x->data[i] = 0;
}

__host__ __device__ __forceinline__ void init_bigint_from_u64(BigInt *a, uint64_t val) {
    a->data[0] = (uint32_t)(val & 0xFFFFFFFF);
    a->data[1] = (uint32_t)(val >> 32);
    for (int i = 2; i < BIGINT_WORDS; i++) {
        a->data[i] = 0;
    }
}

__host__ __device__ __forceinline__ void copy_bigint(BigInt *dest, const BigInt *src) {
    for (int i = 0; i < BIGINT_WORDS; i++) {
        dest->data[i] = src->data[i];
    }
}

__host__ __device__ __forceinline__ int compare_bigint(const BigInt *a, const BigInt *b) {
    for (int i = BIGINT_WORDS - 1; i >= 0; i--) {
        if (a->data[i] > b->data[i]) return 1;
        if (a->data[i] < b->data[i]) return -1;
    }
    return 0;
}

__host__ __device__ __forceinline__ bool is_zero(const BigInt *a) {
    for (int i = 0; i < BIGINT_WORDS; i++) {
        if (a->data[i]) return false;
    }
    return true;
}

__host__ __device__ __forceinline__ int get_bit(const BigInt *a, int i) {
    int word_idx = i >> 5;
    int bit_idx = i & 31;
    if (word_idx >= BIGINT_WORDS) return 0;
    return (a->data[word_idx] >> bit_idx) & 1;
}

__host__ __device__ __forceinline__ void add_scalar_to_bigint(BigInt *res, const BigInt *a, unsigned long long scalar) {
    uint64_t carry = scalar;
    for (int i = 0; i < BIGINT_WORDS; ++i) {
        uint64_t sum = (uint64_t)a->data[i] + (carry & 0xFFFFFFFFULL);
        res->data[i] = (uint32_t)sum;
        carry = (sum >> 32) + (carry >> 32);
    }
}

__host__ __device__ __forceinline__ void ptx_u256Add(BigInt *res, const BigInt *a, const BigInt *b) {
    uint64_t carry = 0;
    for (int i = 0; i < BIGINT_WORDS; ++i) {
        uint64_t sum = (uint64_t)a->data[i] + b->data[i] + carry;
        res->data[i] = (uint32_t)sum;
        carry = (sum >> 32);
    }
}

__host__ __device__ __forceinline__ void bigint_add(BigInt *res, const BigInt *a, const BigInt *b) {
    ptx_u256Add(res, a, b);
}

__host__ __device__ __forceinline__ void ptx_u256Sub(BigInt *res, const BigInt *a, const BigInt *b) {
    uint32_t borrow = 0;
    for (int i = 0; i < BIGINT_WORDS; ++i) {
        uint64_t tmp = (uint64_t)a->data[i] - borrow;
        borrow = tmp < b->data[i] ? 1u : 0u;
        res->data[i] = (uint32_t)(tmp - b->data[i]);
    }
}

__host__ __device__ __forceinline__ void bigint_mul_uint32(BigInt *res, const BigInt *a, uint32_t b_val) {
    uint64_t carry = 0;
    for (int i = 0; i < BIGINT_WORDS; ++i) {
        uint64_t product = (uint64_t)a->data[i] * b_val + carry;
        res->data[i] = (uint32_t)product;
        carry = product >> 32;
    }
}

__device__ __forceinline__ void multiply_bigint_by_const(const BigInt *a, uint32_t c, uint32_t result[9]) {
    uint64_t carry = 0;
    for (int i = 0; i < BIGINT_WORDS; i++) {
        uint64_t prod = (uint64_t)a->data[i] * c + carry;
        result[i] = (uint32_t)prod;
        carry = prod >> 32;
    }
    result[8] = (uint32_t)carry;
}

__device__ __forceinline__ void shift_left_word(const BigInt *a, uint32_t result[9]) {
    result[0] = 0;
    for (int i = 0; i < BIGINT_WORDS; i++) {
        result[i+1] = a->data[i];
    }
}

__device__ __forceinline__ void add_9word(uint32_t r[9], const uint32_t addend[9]) {
    uint64_t carry = 0;
    for (int i = 0; i < 9; i++) {
        uint64_t sum = (uint64_t)r[i] + addend[i] + carry;
        r[i] = (uint32_t)sum;
        carry = sum >> 32;
    }
}

__device__ __forceinline__ void convert_9word_to_bigint(const uint32_t r[9], BigInt *res) {
    for (int i = 0; i < BIGINT_WORDS; i++) {
        res->data[i] = r[i];
    }
}

__device__ __forceinline__ void mul_mod_device(BigInt *res, const BigInt *a, const BigInt *b) {
    uint32_t prod[2 * BIGINT_WORDS] = {0};
    for (int i = 0; i < BIGINT_WORDS; i++) {
        uint64_t carry = 0;
        for (int j = 0; j < BIGINT_WORDS; j++) {
            uint64_t tmp = (uint64_t)prod[i + j] + (uint64_t)a->data[i] * b->data[j] + carry;
            prod[i + j] = (uint32_t)tmp;
            carry = tmp >> 32;
        }
        prod[i + BIGINT_WORDS] += (uint32_t)carry;
    }

    BigInt L, H;
    for (int i = 0; i < BIGINT_WORDS; i++) {
        L.data[i] = prod[i];
        H.data[i] = prod[i + BIGINT_WORDS];
    }

    uint32_t Rext[9] = {0};
    for (int i = 0; i < BIGINT_WORDS; i++) Rext[i] = L.data[i];
    Rext[8] = 0;

    uint32_t H977[9] = {0};
    multiply_bigint_by_const(&H, 977, H977);
    add_9word(Rext, H977);

    uint32_t Hshift[9] = {0};
    shift_left_word(&H, Hshift);
    add_9word(Rext, Hshift);

    if (Rext[8]) {
        uint32_t extra[9] = {0};
        BigInt extraBI;
        init_bigint(&extraBI, Rext[8]);
        Rext[8] = 0;

        uint32_t extra977[9] = {0}, extraShift[9] = {0};
        multiply_bigint_by_const(&extraBI, 977, extra977);
        shift_left_word(&extraBI, extraShift);

        for (int i = 0; i < 9; i++) extra[i] = extra977[i];
        add_9word(extra, extraShift);
        add_9word(Rext, extra);
    }

    BigInt R_temp;
    convert_9word_to_bigint(Rext, &R_temp);

    if (Rext[8] || compare_bigint(&R_temp, &const_p) >= 0) {
        ptx_u256Sub(&R_temp, &R_temp, &const_p);
    }
    if (compare_bigint(&R_temp, &const_p) >= 0) {
        ptx_u256Sub(&R_temp, &R_temp, &const_p);
    }

    copy_bigint(res, &R_temp);
}

__device__ __forceinline__ void sub_mod_device(BigInt *res, const BigInt *a, const BigInt *b) {
    BigInt temp;
    if (compare_bigint(a, b) < 0) {
         BigInt sum;
         ptx_u256Add(&sum, a, &const_p);
         ptx_u256Sub(&temp, &sum, b);
    } else {
         ptx_u256Sub(&temp, a, b);
    }
    copy_bigint(res, &temp);
}

__device__ __forceinline__ void scalar_mod_n(BigInt *res, const BigInt *a) {
    if (compare_bigint(a, &const_n) >= 0) {
        ptx_u256Sub(res, a, &const_n);
    } else {
        copy_bigint(res, a);
    }
}

__device__ __forceinline__ void add_mod_device(BigInt *res, const BigInt *a, const BigInt *b) {
    BigInt sum_ab;
    uint64_t carry = 0;
    for (int i = 0; i < BIGINT_WORDS; ++i) {
         uint64_t word_sum = (uint64_t)a->data[i] + b->data[i] + carry;
         sum_ab.data[i] = (uint32_t)word_sum;
         carry = word_sum >> 32;
    }
    if (carry || compare_bigint(&sum_ab, &const_p) >= 0) {
        ptx_u256Sub(res, &sum_ab, &const_p);
    } else {
        copy_bigint(res, &sum_ab);
    }
}

__device__ void modexp(BigInt *res, const BigInt *base, const BigInt *exp) {
    BigInt result;
    init_bigint(&result, 1);
    BigInt b;
    copy_bigint(&b, base);
    for (int i = 0; i < 256; i++) {
         if (get_bit(exp, i)) {
              mul_mod_device(&result, &result, &b);
         }
         mul_mod_device(&b, &b, &b);
    }
    copy_bigint(res, &result);
}

__device__ void mod_inverse(BigInt *res, const BigInt *a) {
    if (is_zero(a)) {
        init_bigint(res, 0);
        return;
    }
    BigInt p_minus_2, two;
    init_bigint(&two, 2);
    ptx_u256Sub(&p_minus_2, &const_p, &two);
    modexp(res, a, &p_minus_2);
}

__device__ __forceinline__ void point_set_infinity_jac(ECPointJac *P) {
    P->infinity = true;
}

__device__ __forceinline__ void point_copy_jac(ECPointJac *dest, const ECPointJac *src) {
    copy_bigint(&dest->X, &src->X);
    copy_bigint(&dest->Y, &src->Y);
    copy_bigint(&dest->Z, &src->Z);
    dest->infinity = src->infinity;
}

__device__ void double_point_jac(ECPointJac *R, const ECPointJac *P);
__device__ void add_point_jac(ECPointJac *R, const ECPointJac *P, const ECPointJac *Q);

__device__ void double_point_jac(ECPointJac *R, const ECPointJac *P) {
    if (P->infinity || is_zero(&P->Y)) {
        point_set_infinity_jac(R);
        return;
    }
    BigInt A, B, C, D, X3, Y3, Z3, temp, temp2;
    mul_mod_device(&A, &P->Y, &P->Y);
    mul_mod_device(&temp, &P->X, &A);
    init_bigint(&temp2, 4);
    mul_mod_device(&B, &temp, &temp2);
    mul_mod_device(&temp, &A, &A);
    init_bigint(&temp2, 8);
    mul_mod_device(&C, &temp, &temp2);
    mul_mod_device(&temp, &P->X, &P->X);
    init_bigint(&temp2, 3);
    mul_mod_device(&D, &temp, &temp2);
    BigInt D2, two, twoB;
    mul_mod_device(&D2, &D, &D);
    init_bigint(&two, 2);
    mul_mod_device(&twoB, &B, &two);
    sub_mod_device(&X3, &D2, &twoB);
    sub_mod_device(&temp, &B, &X3);
    mul_mod_device(&temp, &D, &temp);
    sub_mod_device(&Y3, &temp, &C);
    init_bigint(&temp, 2);
    mul_mod_device(&temp, &temp, &P->Y);
    mul_mod_device(&Z3, &temp, &P->Z);
    copy_bigint(&R->X, &X3);
    copy_bigint(&R->Y, &Y3);
    copy_bigint(&R->Z, &Z3);
    R->infinity = false;
}

__device__ void add_point_jac(ECPointJac *R, const ECPointJac *P, const ECPointJac *Q) {
    if (P->infinity) { point_copy_jac(R, Q); return; }
    if (Q->infinity) { point_copy_jac(R, P); return; }

    BigInt Z1Z1, Z2Z2, U1, U2, S1, S2, H, R_big, H2, H3, U1H2, X3, Y3, Z3, temp;
    mul_mod_device(&Z1Z1, &P->Z, &P->Z);
    mul_mod_device(&Z2Z2, &Q->Z, &Q->Z);
    mul_mod_device(&U1, &P->X, &Z2Z2);
    mul_mod_device(&U2, &Q->X, &Z1Z1);
    BigInt Z2_cubed, Z1_cubed;
    mul_mod_device(&temp, &Z2Z2, &Q->Z); copy_bigint(&Z2_cubed, &temp);
    mul_mod_device(&temp, &Z1Z1, &P->Z); copy_bigint(&Z1_cubed, &temp);
    mul_mod_device(&S1, &P->Y, &Z2_cubed);
    mul_mod_device(&S2, &Q->Y, &Z1_cubed);

    if (compare_bigint(&U1, &U2) == 0) {
        if (compare_bigint(&S1, &S2) != 0) {
            point_set_infinity_jac(R);
            return;
        } else {
            double_point_jac(R, P);
            return;
        }
    }
    sub_mod_device(&H, &U2, &U1);
    sub_mod_device(&R_big, &S2, &S1);
    mul_mod_device(&H2, &H, &H);
    mul_mod_device(&H3, &H2, &H);
    mul_mod_device(&U1H2, &U1, &H2);
    BigInt R2, two, twoU1H2;
    mul_mod_device(&R2, &R_big, &R_big);
    init_bigint(&two, 2);
    mul_mod_device(&twoU1H2, &U1H2, &two);
    sub_mod_device(&temp, &R2, &H3);
    sub_mod_device(&X3, &temp, &twoU1H2);
    sub_mod_device(&temp, &U1H2, &X3);
    mul_mod_device(&temp, &R_big, &temp);
    mul_mod_device(&Y3, &S1, &H3);
    sub_mod_device(&Y3, &temp, &Y3);
    mul_mod_device(&temp, &P->Z, &Q->Z);
    mul_mod_device(&Z3, &temp, &H);
    copy_bigint(&R->X, &X3);
    copy_bigint(&R->Y, &Y3);
    copy_bigint(&R->Z, &Z3);
    R->infinity = false;
}

__device__ void jacobian_to_affine(ECPoint *R, const ECPointJac *P) {
    if (P->infinity) {
        R->infinity = true;
        init_bigint(&R->x, 0);
        init_bigint(&R->y, 0);
        return;
    }
    BigInt Zinv, Zinv2, Zinv3;
    mod_inverse(&Zinv, &P->Z);
    mul_mod_device(&Zinv2, &Zinv, &Zinv);
    mul_mod_device(&Zinv3, &Zinv2, &Zinv);
    mul_mod_device(&R->x, &P->X, &Zinv2);
    mul_mod_device(&R->y, &P->Y, &Zinv3);
    R->infinity = false;
}

__device__ void scalar_multiply_jac_device(ECPointJac *result, const ECPointJac *point, const BigInt *scalar) {
    ECPointJac res;
    point_set_infinity_jac(&res);

    int highest_bit = BIGINT_WORDS * 32 - 1;
    for (; highest_bit >= 0; highest_bit--) {
        if (get_bit(scalar, highest_bit)) break;
    }

    if (highest_bit < 0) {
        point_copy_jac(result, &res);
        return;
    }

    ECPointJac p_copy;
    point_copy_jac(&p_copy, point);

    for (int i = highest_bit; i >= 0; i--) {
        double_point_jac(&res, &res);
        if (get_bit(scalar, i)) {
            add_point_jac(&res, &res, &p_copy);
        }
    }
    point_copy_jac(result, &res);
}

// ===================================================================
// PRECOMPUTATION FUNCTIONS
// ===================================================================
extern "C"
__global__ void precompute_G_table_kernel() {
    int idx = threadIdx.x;
    if (idx == 0) {
        ECPointJac current = const_G_jacobian;
        point_copy_jac(&const_G_table[0], &current);  // Simpan G (2^0 * G)

        // Precompute 2^i * G untuk i=1 sampai 255
        for (int i = 1; i < 256; i++) {
            double_point_jac(&current, &current);
            point_copy_jac(&const_G_table[i], &current);
        }
    }
}

__device__ void scalar_multiply_jac_precomputed(ECPointJac *result, const BigInt *scalar) {
    point_set_infinity_jac(result);

    for (int i = 0; i < 256; i++) {
        if (get_bit(scalar, i)) {
            add_point_jac(result, result, &const_G_table[i]);
        }
    }
}

// ===================================================================
// KERNEL UTAMA UNTUK PENCARIAN HASH160
// ===================================================================

// --- Kode dari GPUHash.h ---

__device__ __constant__ uint32_t K[] =
{
    0x428A2F98, 0x71374491, 0xB5C0FBCF, 0xE9B5DBA5,
    0x3956C25B, 0x59F111F1, 0x923F82A4, 0xAB1C5ED5,
    0xD807AA98, 0x12835B01, 0x243185BE, 0x550C7DC3,
    0x72BE5D74, 0x80DEB1FE, 0x9BDC06A7, 0xC19BF174,
    0xE49B69C1, 0xEFBE4786, 0x0FC19DC6, 0x240CA1CC,
    0x2DE92C6F, 0x4A7484AA, 0x5CB0A9DC, 0x76F988DA,
    0x983E5152, 0xA831C66D, 0xB00327C8, 0xBF597FC7,
    0xC6E00BF3, 0xD5A79147, 0x06CA6351, 0x14292967,
    0x27B70A85, 0x2E1B2138, 0x4D2C6DFC, 0x53380D13,
    0x650A7354, 0x766A0ABB, 0x81C2C92E, 0x92722C85,
    0xA2BFE8A1, 0xA81A664B, 0xC24B8B70, 0xC76C51A3,
    0xD192E819, 0xD6990624, 0xF40E3585, 0x106AA070,
    0x19A4C116, 0x1E376C08, 0x2748774C, 0x34B0BCB5,
    0x391C0CB3, 0x4ED8AA4A, 0x5B9CCA4F, 0x682E6FF3,
    0x748F82EE, 0x78A5636F, 0x84C87814, 0x8CC70208,
    0x90BEFFFA, 0xA4506CEB, 0xBEF9A3F7, 0xC67178F2,
};

__device__ __constant__ uint32_t I[] = {
  0x6a09e667ul,
  0xbb67ae85ul,
  0x3c6ef372ul,
  0xa54ff53aul,
  0x510e527ful,
  0x9b05688cul,
  0x1f83d9abul,
  0x5be0cd19ul,
};

#define ROR(x,n) ((x>>n)|(x<<(32-n)))
#define S0(x) (ROR(x,2) ^ ROR(x,13) ^ ROR(x,22))
#define S1(x) (ROR(x,6) ^ ROR(x,11) ^ ROR(x,25))
#define s0(x) (ROR(x,7) ^ ROR(x,18) ^ (x >> 3))
#define s1(x) (ROR(x,17) ^ ROR(x,19) ^ (x >> 10))

#define Maj(x,y,z) ((x & y) | (z & (x | y)))
#define Ch(x,y,z) (z ^ (x & (y ^ z)))

// SHA-256 inner round
#define S2Round(a, b, c, d, e, f, g, h, k, w) \
    t1 = h + S1(e) + Ch(e,f,g) + k + (w); \
    t2 = S0(a) + Maj(a,b,c); \
    d += t1; \
    h = t1 + t2;

// WMIX
#define WMIX() { \
w[0] += s1(w[14]) + w[9] + s0(w[1]);\
w[1] += s1(w[15]) + w[10] + s0(w[2]);\
w[2] += s1(w[0]) + w[11] + s0(w[3]);\
w[3] += s1(w[1]) + w[12] + s0(w[4]);\
w[4] += s1(w[2]) + w[13] + s0(w[5]);\
w[5] += s1(w[3]) + w[14] + s0(w[6]);\
w[6] += s1(w[4]) + w[15] + s0(w[7]);\
w[7] += s1(w[5]) + w[0] + s0(w[8]);\
w[8] += s1(w[6]) + w[1] + s0(w[9]);\
w[9] += s1(w[7]) + w[2] + s0(w[10]);\
w[10] += s1(w[8]) + w[3] + s0(w[11]);\
w[11] += s1(w[9]) + w[4] + s0(w[12]);\
w[12] += s1(w[10]) + w[5] + s0(w[13]);\
w[13] += s1(w[11]) + w[6] + s0(w[14]);\
w[14] += s1(w[12]) + w[7] + s0(w[15]);\
w[15] += s1(w[13]) + w[8] + s0(w[0]);\
}

// ROUND
#define SHA256_RND(k) {\
S2Round(a, b, c, d, e, f, g, h, K[k], w[0]);\
S2Round(h, a, b, c, d, e, f, g, K[k + 1], w[1]);\
S2Round(g, h, a, b, c, d, e, f, K[k + 2], w[2]);\
S2Round(f, g, h, a, b, c, d, e, K[k + 3], w[3]);\
S2Round(e, f, g, h, a, b, c, d, K[k + 4], w[4]);\
S2Round(d, e, f, g, h, a, b, c, K[k + 5], w[5]);\
S2Round(c, d, e, f, g, h, a, b, K[k + 6], w[6]);\
S2Round(b, c, d, e, f, g, h, a, K[k + 7], w[7]);\
S2Round(a, b, c, d, e, f, g, h, K[k + 8], w[8]);\
S2Round(h, a, b, c, d, e, f, g, K[k + 9], w[9]);\
S2Round(g, h, a, b, c, d, e, f, K[k + 10], w[10]);\
S2Round(f, g, h, a, b, c, d, e, K[k + 11], w[11]);\
S2Round(e, f, g, h, a, b, c, d, K[k + 12], w[12]);\
S2Round(d, e, f, g, h, a, b, c, K[k + 13], w[13]);\
S2Round(c, d, e, f, g, h, a, b, K[k + 14], w[14]);\
S2Round(b, c, d, e, f, g, h, a, K[k + 15], w[15]);\
}

#define bswap32(v) __byte_perm(v, 0, 0x0123)

// Initialise state
__device__ void SHA256Initialize(uint32_t s[8]) {
#pragma unroll 8
  for (int i = 0; i < 8; i++)
    s[i] = I[i];
}

#define DEF(x,y) uint32_t x = s[y]

// Perform SHA-256 transformations, process 64-byte chunks
__device__ void SHA256Transform(uint32_t s[8],uint32_t* w) {

  uint32_t t1;
  uint32_t t2;

  DEF(a, 0);
  DEF(b, 1);
  DEF(c, 2);
  DEF(d, 3);
  DEF(e, 4);
  DEF(f, 5);
  DEF(g, 6);
  DEF(h, 7);

  SHA256_RND(0);
  WMIX();
  SHA256_RND(16);
  WMIX();
  SHA256_RND(32);
  WMIX();
  SHA256_RND(48);

  s[0] += a;
  s[1] += b;
  s[2] += c;
  s[3] += d;
  s[4] += e;
  s[5] += f;
  s[6] += g;
  s[7] += h;

}

// ---------------------------------------------------------------------------------
// RIPEMD160
// ---------------------------------------------------------------------------------
__device__ __constant__ uint64_t ripemd160_sizedesc_32 = 32 << 3;

__device__ void RIPEMD160Initialize(uint32_t s[5]) {

  s[0] = 0x67452301ul;
  s[1] = 0xEFCDAB89ul;
  s[2] = 0x98BADCFEul;
  s[3] = 0x10325476ul;
  s[4] = 0xC3D2E1F0ul;

}

#define ROL(x,n) ((x>>(32-n))|(x<<n))
#define f1(x, y, z) (x ^ y ^ z)
#define f2(x, y, z) ((x & y) | (~x & z))
#define f3(x, y, z) ((x | ~y) ^ z)
#define f4(x, y, z) ((x & z) | (~z & y))
#define f5(x, y, z) (x ^ (y | ~z))

#define RPRound(a,b,c,d,e,f,x,k,r) \
  u = a + f + x + k; \
  a = ROL(u, r) + e; \
  c = ROL(c, 10);

#define R11(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f1(b, c, d), x, 0, r)
#define R21(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f2(b, c, d), x, 0x5A827999ul, r)
#define R31(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f3(b, c, d), x, 0x6ED9EBA1ul, r)
#define R41(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f4(b, c, d), x, 0x8F1BBCDCul, r)
#define R51(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f5(b, c, d), x, 0xA953FD4Eul, r)
#define R12(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f5(b, c, d), x, 0x50A28BE6ul, r)
#define R22(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f4(b, c, d), x, 0x5C4DD124ul, r)
#define R32(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f3(b, c, d), x, 0x6D703EF3ul, r)
#define R42(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f2(b, c, d), x, 0x7A6D76E9ul, r)
#define R52(a,b,c,d,e,x,r) RPRound(a, b, c, d, e, f1(b, c, d), x, 0, r)

/** Perform a RIPEMD-160 transformation, processing a 64-byte chunk. */
__device__ void RIPEMD160Transform(uint32_t s[5],uint32_t* w) {

  uint32_t u;
  uint32_t a1 = s[0], b1 = s[1], c1 = s[2], d1 = s[3], e1 = s[4];
  uint32_t a2 = a1, b2 = b1, c2 = c1, d2 = d1, e2 = e1;

  R11(a1, b1, c1, d1, e1, w[0], 11);
  R12(a2, b2, c2, d2, e2, w[5], 8);
  R11(e1, a1, b1, c1, d1, w[1], 14);
  R12(e2, a2, b2, c2, d2, w[14], 9);
  R11(d1, e1, a1, b1, c1, w[2], 15);
  R12(d2, e2, a2, b2, c2, w[7], 9);
  R11(c1, d1, e1, a1, b1, w[3], 12);
  R12(c2, d2, e2, a2, b2, w[0], 11);
  R11(b1, c1, d1, e1, a1, w[4], 5);
  R12(b2, c2, d2, e2, a2, w[9], 13);
  R11(a1, b1, c1, d1, e1, w[5], 8);
  R12(a2, b2, c2, d2, e2, w[2], 15);
  R11(e1, a1, b1, c1, d1, w[6], 7);
  R12(e2, a2, b2, c2, d2, w[11], 15);
  R11(d1, e1, a1, b1, c1, w[7], 9);
  R12(d2, e2, a2, b2, c2, w[4], 5);
  R11(c1, d1, e1, a1, b1, w[8], 11);
  R12(c2, d2, e2, a2, b2, w[13], 7);
  R11(b1, c1, d1, e1, a1, w[9], 13);
  R12(b2, c2, d2, e2, a2, w[6], 7);
  R11(a1, b1, c1, d1, e1, w[10], 14);
  R12(a2, b2, c2, d2, e2, w[15], 8);
  R11(e1, a1, b1, c1, d1, w[11], 15);
  R12(e2, a2, b2, c2, d2, w[8], 11);
  R11(d1, e1, a1, b1, c1, w[12], 6);
  R12(d2, e2, a2, b2, c2, w[1], 14);
  R11(c1, d1, e1, a1, b1, w[13], 7);
  R12(c2, d2, e2, a2, b2, w[10], 14);
  R11(b1, c1, d1, e1, a1, w[14], 9);
  R12(b2, c2, d2, e2, a2, w[3], 12);
  R11(a1, b1, c1, d1, e1, w[15], 8);
  R12(a2, b2, c2, d2, e2, w[12], 6);

  R21(e1, a1, b1, c1, d1, w[7], 7);
  R22(e2, a2, b2, c2, d2, w[6], 9);
  R21(d1, e1, a1, b1, c1, w[4], 6);
  R22(d2, e2, a2, b2, c2, w[11], 13);
  R21(c1, d1, e1, a1, b1, w[13], 8);
  R22(c2, d2, e2, a2, b2, w[3], 15);
  R21(b1, c1, d1, e1, a1, w[1], 13);
  R22(b2, c2, d2, e2, a2, w[7], 7);
  R21(a1, b1, c1, d1, e1, w[10], 11);
  R22(a2, b2, c2, d2, e2, w[0], 12);
  R21(e1, a1, b1, c1, d1, w[6], 9);
  R22(e2, a2, b2, c2, d2, w[13], 8);
  R21(d1, e1, a1, b1, c1, w[15], 7);
  R22(d2, e2, a2, b2, c2, w[5], 9);
  R21(c1, d1, e1, a1, b1, w[3], 15);
  R22(c2, d2, e2, a2, b2, w[10], 11);
  R21(b1, c1, d1, e1, a1, w[12], 7);
  R22(b2, c2, d2, e2, a2, w[14], 7);
  R21(a1, b1, c1, d1, e1, w[0], 12);
  R22(a2, b2, c2, d2, e2, w[15], 7);
  R21(e1, a1, b1, c1, d1, w[9], 15);
  R22(e2, a2, b2, c2, d2, w[8], 12);
  R21(d1, e1, a1, b1, c1, w[5], 9);
  R22(d2, e2, a2, b2, c2, w[12], 7);
  R21(c1, d1, e1, a1, b1, w[2], 11);
  R22(c2, d2, e2, a2, b2, w[4], 6);
  R21(b1, c1, d1, e1, a1, w[14], 7);
  R22(b2, c2, d2, e2, a2, w[9], 15);
  R21(a1, b1, c1, d1, e1, w[11], 13);
  R22(a2, b2, c2, d2, e2, w[1], 13);
  R21(e1, a1, b1, c1, d1, w[8], 12);
  R22(e2, a2, b2, c2, d2, w[2], 11);

  R31(d1, e1, a1, b1, c1, w[3], 11);
  R32(d2, e2, a2, b2, c2, w[15], 9);
  R31(c1, d1, e1, a1, b1, w[10], 13);
  R32(c2, d2, e2, a2, b2, w[5], 7);
  R31(b1, c1, d1, e1, a1, w[14], 6);
  R32(b2, c2, d2, e2, a2, w[1], 15);
  R31(a1, b1, c1, d1, e1, w[4], 7);
  R32(a2, b2, c2, d2, e2, w[3], 11);
  R31(e1, a1, b1, c1, d1, w[9], 14);
  R32(e2, a2, b2, c2, d2, w[7], 8);
  R31(d1, e1, a1, b1, c1, w[15], 9);
  R32(d2, e2, a2, b2, c2, w[14], 6);
  R31(c1, d1, e1, a1, b1, w[8], 13);
  R32(c2, d2, e2, a2, b2, w[6], 6);
  R31(b1, c1, d1, e1, a1, w[1], 15);
  R32(b2, c2, d2, e2, a2, w[9], 14);
  R31(a1, b1, c1, d1, e1, w[2], 14);
  R32(a2, b2, c2, d2, e2, w[11], 12);
  R31(e1, a1, b1, c1, d1, w[7], 8);
  R32(e2, a2, b2, c2, d2, w[8], 13);
  R31(d1, e1, a1, b1, c1, w[0], 13);
  R32(d2, e2, a2, b2, c2, w[12], 5);
  R31(c1, d1, e1, a1, b1, w[6], 6);
  R32(c2, d2, e2, a2, b2, w[2], 14);
  R31(b1, c1, d1, e1, a1, w[13], 5);
  R32(b2, c2, d2, e2, a2, w[10], 13);
  R31(a1, b1, c1, d1, e1, w[11], 12);
  R32(a2, b2, c2, d2, e2, w[0], 13);
  R31(e1, a1, b1, c1, d1, w[5], 7);
  R32(e2, a2, b2, c2, d2, w[4], 7);
  R31(d1, e1, a1, b1, c1, w[12], 5);
  R32(d2, e2, a2, b2, c2, w[13], 5);

  R41(c1, d1, e1, a1, b1, w[1], 11);
  R42(c2, d2, e2, a2, b2, w[8], 15);
  R41(b1, c1, d1, e1, a1, w[9], 12);
  R42(b2, c2, d2, e2, a2, w[6], 5);
  R41(a1, b1, c1, d1, e1, w[11], 14);
  R42(a2, b2, c2, d2, e2, w[4], 8);
  R41(e1, a1, b1, c1, d1, w[10], 15);
  R42(e2, a2, b2, c2, d2, w[1], 11);
  R41(d1, e1, a1, b1, c1, w[0], 14);
  R42(d2, e2, a2, b2, c2, w[3], 14);
  R41(c1, d1, e1, a1, b1, w[8], 15);
  R42(c2, d2, e2, a2, b2, w[11], 14);
  R41(b1, c1, d1, e1, a1, w[12], 9);
  R42(b2, c2, d2, e2, a2, w[15], 6);
  R41(a1, b1, c1, d1, e1, w[4], 8);
  R42(a2, b2, c2, d2, e2, w[0], 14);
  R41(e1, a1, b1, c1, d1, w[13], 9);
  R42(e2, a2, b2, c2, d2, w[5], 6);
  R41(d1, e1, a1, b1, c1, w[3], 14);
  R42(d2, e2, a2, b2, c2, w[12], 9);
  R41(c1, d1, e1, a1, b1, w[7], 5);
  R42(c2, d2, e2, a2, b2, w[2], 12);
  R41(b1, c1, d1, e1, a1, w[15], 6);
  R42(b2, c2, d2, e2, a2, w[13], 9);
  R41(a1, b1, c1, d1, e1, w[14], 8);
  R42(a2, b2, c2, d2, e2, w[9], 12);
  R41(e1, a1, b1, c1, d1, w[5], 6);
  R42(e2, a2, b2, c2, d2, w[7], 5);
  R41(d1, e1, a1, b1, c1, w[6], 5);
  R42(d2, e2, a2, b2, c2, w[10], 15);
  R41(c1, d1, e1, a1, b1, w[2], 12);
  R42(c2, d2, e2, a2, b2, w[14], 8);

  R51(b1, c1, d1, e1, a1, w[4], 9);
  R52(b2, c2, d2, e2, a2, w[12], 8);
  R51(a1, b1, c1, d1, e1, w[0], 15);
  R52(a2, b2, c2, d2, e2, w[15], 5);
  R51(e1, a1, b1, c1, d1, w[5], 5);
  R52(e2, a2, b2, c2, d2, w[10], 12);
  R51(d1, e1, a1, b1, c1, w[9], 11);
  R52(d2, e2, a2, b2, c2, w[4], 9);
  R51(c1, d1, e1, a1, b1, w[7], 6);
  R52(c2, d2, e2, a2, b2, w[1], 12);
  R51(b1, c1, d1, e1, a1, w[12], 8);
  R52(b2, c2, d2, e2, a2, w[5], 5);
  R51(a1, b1, c1, d1, e1, w[2], 13);
  R52(a2, b2, c2, d2, e2, w[8], 14);
  R51(e1, a1, b1, c1, d1, w[10], 12);
  R52(e2, a2, b2, c2, d2, w[7], 6);
  R51(d1, e1, a1, b1, c1, w[14], 5);
  R52(d2, e2, a2, b2, c2, w[6], 8);
  R51(c1, d1, e1, a1, b1, w[1], 12);
  R52(c2, d2, e2, a2, b2, w[2], 13);
  R51(b1, c1, d1, e1, a1, w[3], 13);
  R52(b2, c2, d2, e2, a2, w[13], 6);
  R51(a1, b1, c1, d1, e1, w[8], 14);
  R52(a2, b2, c2, d2, e2, w[14], 5);
  R51(e1, a1, b1, c1, d1, w[11], 11);
  R52(e2, a2, b2, c2, d2, w[0], 15);
  R51(d1, e1, a1, b1, c1, w[6], 8);
  R52(d2, e2, a2, b2, c2, w[3], 13);
  R51(c1, d1, e1, a1, b1, w[15], 5);
  R52(c2, d2, e2, a2, b2, w[9], 11);
  R51(b1, c1, d1, e1, a1, w[13], 6);
  R52(b2, c2, d2, e2, a2, w[11], 11);

  uint32_t t = s[0];
  s[0] = s[1] + c1 + d2;
  s[1] = s[2] + d1 + e2;
  s[2] = s[3] + e1 + a2;
  s[3] = s[4] + a1 + b2;
  s[4] = t + b1 + c2;
}

// ---------------------------------------------------------------------------------
// Key encoding
// ---------------------------------------------------------------------------------

__device__ __noinline__ void _GetHash160Comp(uint64_t *x, uint8_t isOdd, uint8_t *hash) {

  uint32_t *x32 = (uint32_t *)(x);
  uint32_t publicKeyBytes[16];
  uint32_t s[16];

  // Compressed public key
  publicKeyBytes[0] = __byte_perm(x32[7], 0x2 + isOdd, 0x4321);
  publicKeyBytes[1] = __byte_perm(x32[7], x32[6], 0x0765);
  publicKeyBytes[2] = __byte_perm(x32[6], x32[5], 0x0765);
  publicKeyBytes[3] = __byte_perm(x32[5], x32[4], 0x0765);
  publicKeyBytes[4] = __byte_perm(x32[4], x32[3], 0x0765);
  publicKeyBytes[5] = __byte_perm(x32[3], x32[2], 0x0765);
  publicKeyBytes[6] = __byte_perm(x32[2], x32[1], 0x0765);
  publicKeyBytes[7] = __byte_perm(x32[1], x32[0], 0x0765);
  publicKeyBytes[8] = __byte_perm(x32[0], 0x80, 0x0456);
  publicKeyBytes[9] = 0;
  publicKeyBytes[10] = 0;
  publicKeyBytes[11] = 0;
  publicKeyBytes[12] = 0;
  publicKeyBytes[13] = 0;
  publicKeyBytes[14] = 0;
  publicKeyBytes[15] = 0x108;

  SHA256Initialize(s);
  SHA256Transform(s, publicKeyBytes);

#pragma unroll 8
  for (int i = 0; i < 8; i++)
    s[i] = bswap32(s[i]);

  *(uint64_t *)(s + 8) = 0x80ULL;
  *(uint64_t *)(s + 10) = 0ULL;
  *(uint64_t *)(s + 12) = 0ULL;
  *(uint64_t *)(s + 14) = ripemd160_sizedesc_32;

  RIPEMD160Initialize((uint32_t *)hash);
  RIPEMD160Transform((uint32_t *)hash, s);

}

__device__ __noinline__ void _GetHash160CompSym(uint64_t *x, uint8_t *h1, uint8_t *h2) {

  uint32_t *x32 = (uint32_t *)(x);
  uint32_t publicKeyBytes[16];
  uint32_t publicKeyBytes2[16];
  uint32_t s[16];

  // Compressed public key

  // Even
  publicKeyBytes[0] = __byte_perm(x32[7], 0x2, 0x4321);
  publicKeyBytes[1] = __byte_perm(x32[7], x32[6], 0x0765);
  publicKeyBytes[2] = __byte_perm(x32[6], x32[5], 0x0765);
  publicKeyBytes[3] = __byte_perm(x32[5], x32[4], 0x0765);
  publicKeyBytes[4] = __byte_perm(x32[4], x32[3], 0x0765);
  publicKeyBytes[5] = __byte_perm(x32[3], x32[2], 0x0765);
  publicKeyBytes[6] = __byte_perm(x32[2], x32[1], 0x0765);
  publicKeyBytes[7] = __byte_perm(x32[1], x32[0], 0x0765);
  publicKeyBytes[8] = __byte_perm(x32[0], 0x80, 0x0456);
  publicKeyBytes[9] = 0;
  publicKeyBytes[10] = 0;
  publicKeyBytes[11] = 0;
  publicKeyBytes[12] = 0;
  publicKeyBytes[13] = 0;
  publicKeyBytes[14] = 0;
  publicKeyBytes[15] = 0x108;

  // Odd
  publicKeyBytes2[0] = __byte_perm(x32[7], 0x3, 0x4321);
  publicKeyBytes2[1] = publicKeyBytes[1];
  *(uint64_t *)(&publicKeyBytes2[2]) = *(uint64_t *)(&publicKeyBytes[2]);
  *(uint64_t *)(&publicKeyBytes2[4]) = *(uint64_t *)(&publicKeyBytes[4]);
  *(uint64_t *)(&publicKeyBytes2[6]) = *(uint64_t *)(&publicKeyBytes[6]);
  *(uint64_t *)(&publicKeyBytes2[8]) = *(uint64_t *)(&publicKeyBytes[8]);
  *(uint64_t *)(&publicKeyBytes2[10]) = *(uint64_t *)(&publicKeyBytes[10]);
  *(uint64_t *)(&publicKeyBytes2[12]) = *(uint64_t *)(&publicKeyBytes[12]);
  *(uint64_t *)(&publicKeyBytes2[14]) = *(uint64_t *)(&publicKeyBytes[14]);

  SHA256Initialize(s);
  SHA256Transform(s, publicKeyBytes);

#pragma unroll 8
  for (int i = 0; i < 8; i++)
    s[i] = bswap32(s[i]);

  *(uint64_t *)(s + 8) = 0x80ULL;
  *(uint64_t *)(s + 10) = 0ULL;
  *(uint64_t *)(s + 12) = 0ULL;
  *(uint64_t *)(s + 14) = ripemd160_sizedesc_32;

  RIPEMD160Initialize((uint32_t *)h1);
  RIPEMD160Transform((uint32_t *)h1, s);

  SHA256Initialize(s);
  SHA256Transform(s, publicKeyBytes2);

#pragma unroll 8
  for (int i = 0; i < 8; i++)
    s[i] = bswap32(s[i]);

  RIPEMD160Initialize((uint32_t *)h2);
  RIPEMD160Transform((uint32_t *)h2, s);

}

__device__ __noinline__ void _GetHash160(uint64_t *x, uint64_t *y, uint8_t *hash) {

  uint32_t *x32 = (uint32_t *)(x);
  uint32_t *y32 = (uint32_t *)(y);
  uint32_t publicKeyBytes[32];
  uint32_t s[16];

  // Uncompressed public key
  publicKeyBytes[0] = __byte_perm(x32[7], 0x04, 0x4321);
  publicKeyBytes[1] = __byte_perm(x32[7], x32[6], 0x0765);
  publicKeyBytes[2] = __byte_perm(x32[6], x32[5], 0x0765);
  publicKeyBytes[3] = __byte_perm(x32[5], x32[4], 0x0765);
  publicKeyBytes[4] = __byte_perm(x32[4], x32[3], 0x0765);
  publicKeyBytes[5] = __byte_perm(x32[3], x32[2], 0x0765);
  publicKeyBytes[6] = __byte_perm(x32[2], x32[1], 0x0765);
  publicKeyBytes[7] = __byte_perm(x32[1], x32[0], 0x0765);
  publicKeyBytes[8] = __byte_perm(x32[0], y32[7], 0x0765);
  publicKeyBytes[9] = __byte_perm(y32[7], y32[6], 0x0765);
  publicKeyBytes[10] = __byte_perm(y32[6], y32[5], 0x0765);
  publicKeyBytes[11] = __byte_perm(y32[5], y32[4], 0x0765);
  publicKeyBytes[12] = __byte_perm(y32[4], y32[3], 0x0765);
  publicKeyBytes[13] = __byte_perm(y32[3], y32[2], 0x0765);
  publicKeyBytes[14] = __byte_perm(y32[2], y32[1], 0x0765);
  publicKeyBytes[15] = __byte_perm(y32[1], y32[0], 0x0765);
  publicKeyBytes[16] = __byte_perm(y32[0], 0x80, 0x0456);
  publicKeyBytes[17] = 0;
  publicKeyBytes[18] = 0;
  publicKeyBytes[19] = 0;
  publicKeyBytes[20] = 0;
  publicKeyBytes[21] = 0;
  publicKeyBytes[22] = 0;
  publicKeyBytes[23] = 0;
  publicKeyBytes[24] = 0;
  publicKeyBytes[25] = 0;
  publicKeyBytes[26] = 0;
  publicKeyBytes[27] = 0;
  publicKeyBytes[28] = 0;
  publicKeyBytes[29] = 0;
  publicKeyBytes[30] = 0;
  publicKeyBytes[31] = 0x208;

  SHA256Initialize(s);
  SHA256Transform(s, publicKeyBytes);
  SHA256Transform(s, publicKeyBytes + 16);

#pragma unroll 8
  for (int i = 0; i < 8; i++)
    s[i] = bswap32(s[i]);

  *(uint64_t *)(s + 8) = 0x80ULL;
  *(uint64_t *)(s + 10) = 0ULL;
  *(uint64_t *)(s + 12) = 0ULL;
  *(uint64_t *)(s + 14) = ripemd160_sizedesc_32;

  RIPEMD160Initialize((uint32_t *)hash);
  RIPEMD160Transform((uint32_t *)hash, s);

}

__device__ inline bool device_memcmp(const void *s1, const void *s2, size_t n) {
    const uint8_t *p1 = (const uint8_t *)s1;
    const uint8_t *p2 = (const uint8_t *)s2;
    for (size_t i = 0; i < n; ++i) {
        if (p1[i] != p2[i]) return false;
    }
    return true;
}

// ===================================================================
// KERNEL UTAMA YANG DIOPTIMISASI UNTUK PENCARIAN HASH160
// ===================================================================

extern "C"
__global__ void find_hash_kernel_optimized(
    const BigInt* start_key,
    unsigned long long keys_per_launch,
    const BigInt* step,
    const uint8_t* d_targets,
    int num_targets,
    BigInt* d_result,
    int* d_found_flag
) {
    unsigned long long idx = (unsigned long long)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= keys_per_launch || *d_found_flag) return;

    // 1. HITUNG PRIVATE KEY: start_key + step * idx
    BigInt current_priv;
    BigInt priv_idx_mul_step;

    // Kalikan step dengan idx
    bigint_mul_uint32(&priv_idx_mul_step, step, (uint32_t)idx);

    // Tambahkan ke start_key
    ptx_u256Add(&current_priv, start_key, &priv_idx_mul_step);

    // Modulo n (curve order)
    scalar_mod_n(&current_priv, &current_priv);

    // 2. PUBLIC KEY DENGAN PRECOMPUTED MULTIPLICATION
    ECPointJac result_jac;
    scalar_multiply_jac_precomputed(&result_jac, &current_priv);

    // 3. KONVERSI KE AFFINE COORDINATES
    ECPoint public_key;
    jacobian_to_affine(&public_key, &result_jac);

    if (public_key.infinity) return;

    // 4. HASH160 COMPUTATION
    uint8_t final_hash160[20];
    uint8_t is_odd = public_key.y.data[0] & 1;
    _GetHash160Comp((uint64_t*)public_key.x.data, is_odd, final_hash160);

    // 5. PERBANDINGAN DENGAN TARGET HASH160
    for (int i = 0; i < num_targets; i++) {
        if (device_memcmp(final_hash160, &d_targets[i * 20], 20)) {
            if (atomicCAS(d_found_flag, 0, 1) == 0) {
                copy_bigint(d_result, &current_priv);
            }
            return;
        }
    }
}

extern "C"
__global__ void find_hash_kernel(
    const BigInt* start_key,
    unsigned long long keys_per_launch,
    const BigInt* step,
    const uint8_t* d_targets,
    int num_targets,
    BigInt* d_result,
    int* d_found_flag
) {
    unsigned long long idx = (unsigned long long)blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= keys_per_launch || *d_found_flag) return;

    // 1. HITUNG PRIVATE KEY: start_key + step * idx
    BigInt priv_idx_mul_step;
    bigint_mul_uint32(&priv_idx_mul_step, step, (uint32_t)idx);

    BigInt current_priv;
    ptx_u256Add(&current_priv, start_key, &priv_idx_mul_step);
    scalar_mod_n(&current_priv, &current_priv);

    // 2. PRIVATE KEY -> PUBLIC KEY (tanpa precomputation)
    ECPointJac result_jac;
    scalar_multiply_jac_device(&result_jac, &const_G_jacobian, &current_priv);

    // 3. KONVERSI KE AFFINE
    ECPoint public_key;
    jacobian_to_affine(&public_key, &result_jac);
    if (public_key.infinity) return;

    // 4. HASH160 COMPUTATION
    uint8_t final_hash160[20];
    uint8_t is_odd = public_key.y.data[0] & 1;
    _GetHash160Comp((uint64_t*)public_key.x.data, is_odd, final_hash160);

    // 5. PERBANDINGAN DENGAN TARGET HASH160
    for (int i = 0; i < num_targets; i++) {
        if (device_memcmp(final_hash160, &d_targets[i * 20], 20)) {
            if (atomicCAS(d_found_flag, 0, 1) == 0) {
                copy_bigint(d_result, &current_priv);
            }
            return;
        }
    }
}

// ===================================================================
// KERNEL TAMBAHAN UNTUK KOMPATIBILITAS
// ===================================================================

__global__ void private_to_public_key_batch_kernel(
    const BigInt *d_private_keys,
    ECPoint *d_public_keys,
    int num_keys)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_keys) return;

    BigInt priv;
    copy_bigint(&priv, &d_private_keys[idx]);
    scalar_mod_n(&priv, &priv);

    ECPointJac result_jac;
    scalar_multiply_jac_device(&result_jac, &const_G_jacobian, &priv);

    ECPoint public_key;
    jacobian_to_affine(&public_key, &result_jac);

    d_public_keys[idx] = public_key;
}
